/* * How to compile (assume cuda is installed at /usr/local/cuda/)
* nvcc -c -I/usr/local/cuda/include getrf_example.cpp
* g++ -fopenmp -o a.out getrf_example.o -L/usr/local/cuda/lib64 -lcusolver -lcudart 
*/

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void printMatrix(int m, int n, const double*A, int lda, const char* name)
{ 
    for(int row = 0 ; row < m ; row++)
    { 
        for(int col = 0 ; col < n ; col++)
        { 
            double Areg = A[row + col*lda]; 
            printf("%s(%d,%d) = %f\n", name, row+1, col+1, Areg); 
        }
    }
}

void loadMatrix(int nrows, int ncols, double *A, int lda, const char *filename)
{
    fstream fin(filename, ios::in);
    if (fin.fail())
    {
        cout << "failed to open file: \"" << filename << "\" for loading" << endl;
        exit(1);
    }

    for (size_t i = 0; i < nrows; i++)
    {
        for (size_t j = 0; j < ncols; j++)
        {
            fin >> A[i + j*lda];
        }
    }

    fin.close();
}

/*    | 1 2 3 | 
* A = | 4 5 6 | 
*     | 7 8 10 | 
* 
* with pivoting: P*A = L*U 
*     | 0 0 1 | 
* P = | 1 0 0 | 
*     | 0 1 0 | 
* 
*     | 1 0 0 |             | 7 8 10 | 
* L = | 0.1429 1 0 |,   U = | 0 0.8571 1.5714 | 
*     | 0.5714 0.5 1 |      | 0 0 -0.5 | 
*/

int main(int argc, char*argv[]) 
{
    const int m = 3; 
    const int lda = m; 
    const int ldb = m;

    double A[lda*m] = { 1.0, 4.0, 7.0, 2.0, 5.0, 8.0, 3.0, 6.0, 10.0}; 
    double B[m] = { 1.0, 2.0, 3.0 }; 
    double X[m]; /* X = A\B */ 
    double LU[lda*m]; /* L and U */ 
    int Ipiv[m]; /* host copy of pivoting sequence */ 
    int info = 0; /* host copy of error info */ 
    double *d_A = NULL; /* device copy of A */ 
    double *d_B = NULL; /* device copy of B */ 

    printf("example of getrf \n"); 

    printf("A = (matlab base-1)\n"); 
    printMatrix(m, m, A, lda, "A"); 
    printf("=====\n"); 
    printf("B = (matlab base-1)\n"); 
    printMatrix(m, 1, B, ldb, "B"); 
    printf("=====\n");

    hipStream_t stream = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess; 
    hipError_t cudaStat2 = hipSuccess; 
    hipError_t cudaStat3 = hipSuccess; 
    hipError_t cudaStat4 = hipSuccess; 

    /* step 1: create cusolver handle, alternatively bind a stream 
       cuSolverDN library was designed to solve dense linear systems */
    hipsolverHandle_t cusolverH = NULL;
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    // If the application performs several small independent computations,
    // or if it makes data transfers in parallel with the computation,
    // CUDA streams can be used to overlap these tasks.
    // cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking); 
    // assert(hipSuccess == cudaStat1); 
    // status = hipsolverSetStream(cusolverH, stream);
    // assert(HIPSOLVER_STATUS_SUCCESS == status);

    /* step 2: allocate device memory and copy A to device */ 
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(double) * lda * m); 
    gpuErrchk(cudaStat1); 
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(double) * m); 
    gpuErrchk(cudaStat2);
    
    cudaStat1 = hipMemcpy(d_A, A, sizeof(double)*lda*m, hipMemcpyHostToDevice); 
    gpuErrchk(cudaStat1); 
    cudaStat2 = hipMemcpy(d_B, B, sizeof(double)*m, hipMemcpyHostToDevice); 
    gpuErrchk(cudaStat2);

    /* step 3: query working space of getrf 
       helper functions calculate the size of work buffers needed
       D = double precision */
    int lwork = 0; /* size of workspace */ 
    status = hipsolverDnDgetrf_bufferSize( cusolverH, m, m, d_A, lda, &lwork); 
    assert(HIPSOLVER_STATUS_SUCCESS == status); 
    double *d_work = NULL; /* device workspace for getrf */ 
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(double)*lwork); 
    assert(hipSuccess == cudaStat1);

    /* step 4: LU factorization */
    int *d_Ipiv = NULL; /* pivoting sequence */ 
    cudaStat3 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m); 
    gpuErrchk(cudaStat3);
    int *d_info = NULL; /* error info */ 
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int)); 
    gpuErrchk(cudaStat4); 

    // perform LU with pivoting
    status = hipsolverDnDgetrf( cusolverH, m, m, d_A, lda, d_work, d_Ipiv, d_info); 
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    cudaStat1 = hipDeviceSynchronize();
    gpuErrchk(cudaStat1);

    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost); 
    gpuErrchk(cudaStat3); 
    if ( 0 > info ){ 
        printf("%d-th parameter is wrong \n", -info); 
        exit(1); 
    } 

    // print pivots
    cudaStat1 = hipMemcpy(Ipiv , d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStat1);
    printf("pivoting sequence, matlab base-1\n"); 
    for(int j = 0 ; j < m ; j++){ 
        printf("Ipiv(%d) = %d\n", j+1, Ipiv[j]); 
    }
    
    printf("L and U = (matlab base-1)\n"); 
    cudaStat2 = hipMemcpy(LU , d_A , sizeof(double)*lda*m, hipMemcpyDeviceToHost);
    gpuErrchk(cudaStat2);
    printMatrix(m, m, LU, lda, "LU");
    printf("=====\n");


    /* * step 5: solve A*X = B 
     *     | 1 |      | -0.3333 | 
     * B = | 2 |, X = | 0.6667 | 
     *     | 3 |      | 0 | 
     */

    int nrhs = 1;
    hipblasOperation_t trans = HIPBLAS_OP_N; //consider normal A, do not transpose
    status = hipsolverDnDgetrs( cusolverH, HIPBLAS_OP_N, m, nrhs, d_A, lda, d_Ipiv, d_B, ldb, d_info); 

    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status); 
    assert(hipSuccess == cudaStat1); 

    // copy back the result
    cudaStat1 = hipMemcpy(X , d_B, sizeof(double)*m, hipMemcpyDeviceToHost); 
    assert(hipSuccess == cudaStat1); 

    printf("X = (matlab base-1)\n"); 
    printMatrix(m, 1, X, ldb, "X");
    printf("=====\n");

     /* free resources */ 
    if (d_A ) hipFree(d_A); 
    if (d_B ) hipFree(d_B); 
    if (d_Ipiv ) hipFree(d_Ipiv); 
    if (d_info ) hipFree(d_info);
    if (d_work ) hipFree(d_work); 
    if (cusolverH ) hipsolverDnDestroy(cusolverH); 
    if (stream ) hipStreamDestroy(stream); 
    hipDeviceReset(); 

    return 0; 
}